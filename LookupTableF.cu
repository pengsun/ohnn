#include "hip/hip_runtime.h"
#include "THC.h"
#include "common.h"
#include <cstdio>
#include <hipsparse.h>


/// Helper
// TODO: make it a template fun and move to a separate file
__global__ void OHNN_CudaLookupTableF_updateOutput_kernel(
		float *inputInd, float *weight, int weightStride, int B, int M, int V, int C,
		float *output, int outputStride)
{
	int iFet = blockIdx.x * blockDim.x + threadIdx.x;
	int iWord = blockIdx.y * blockDim.y + threadIdx.y;
	if (iFet < C && iWord < B*M) {
		int iVocab = (int)(inputInd[iWord] - 1); // C zero base <- lua one base
		int nSrc = iVocab * weightStride + iFet;
		int nDst = iWord * outputStride + iFet;
		output[nDst] = weight[nSrc];
	}
	/*
	printf("blockId = (%d, %d); threadId = (%d, %d), iFet = %d, iWord = %d\n",
			blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, iFet, iWord);
	*/
}


/// Expose
extern "C"
void OHNN_CudaLookupTableF_updateOutput(
		THCState *state,
		// In
        THCudaTensor *input,
        THCudaTensor *weight,
        // Out
        THCudaTensor *output)
{
	DEBUG_PRINT(("in OHNN_CudaLookupTable2_updateOutput\n"));
	THAssert(THCudaTensor_checkGPU(state, 3, input, weight, output));
	// TODO: arg check?

	// input: B, M (,V)
	// weight: V, C
	// output: B, M, C
	int B = THCudaTensor_size(state, input, 0);
	int M = THCudaTensor_size(state, input, 1);
	int V = THCudaTensor_size(state, weight, 0);
	int C = THCudaTensor_size(state, weight, 1);
	DEBUG_PRINT(("B = %d, M = %d, V = %d, C = %d\n", B, M, V, C));

	// prepare data
	THCudaTensor_resize2d(state, output, B*M, C);
	int outputStride = output->stride[0];
	int weightStride = weight->stride[0];
	DEBUG_PRINT(("outputStride = %d\n", outputStride));
	DEBUG_PRINT(("weightStride = %d\n", weightStride));

	// update output
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 grid(DIV_CEIL(C, 32), DIV_CEIL(B*M, 32));
	dim3 block(32, 32);
	OHNN_CudaLookupTableF_updateOutput_kernel<<<grid, block, 0, stream>>>(
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, weight), weightStride,
			B, M, V, C,
			THCudaTensor_data(state, output), outputStride
	);

	// post process
	THCudaTensor_resize3d(state, output, B, M, C);

	// check error
	DEBUG_PRINT(("checking cuda error\n"));
	THCudaCheck(hipGetLastError());
	DEBUG_PRINT(("done, no cuda error\n"));

	DEBUG_PRINT(("leaving OHNN_CudaLookupTable2_updateOutput\n"));
}
