#include "hip/hip_runtime.h"
#include "THC.h"
#include "common.h"
#include <cstdio>
#include <hipsparse.h>


/// Helper
// TODO: make it a template fun and move to a separate file
__global__ void temporal_bow_statck_kernelV1(
		float *input, int B, int M, int p, int padVocabInd,
		float *output)
{
	// a Naive impl
	int nSrc = blockIdx.x * blockDim.x + threadIdx.x;
	if (nSrc >= B*M) return;

	int iBatch = nSrc / M;
	int iWord = nSrc % M;
	for (int i = 0; i < p; ++i) { // scan each word in the window
		int iCurWord = iWord - p/2 + i; // TODO: check?
		int curVocabInd = padVocabInd;
		int nDst = nSrc*p + i;

		if (iCurWord >= 0 && iCurWord < M) { // inside the iBatch sequence
			curVocabInd = (int)input[iBatch*M + iCurWord];
			if (curVocabInd != padVocabInd) { // a normal word
				// scan previous word in the same window, remove any duplicate
				// Warp divergence here?
				for (int k = 1; k <= i; ++k) {
					if (curVocabInd == output[nDst-k]) {
						curVocabInd = padVocabInd;
						break;
					}
				}
			}
		}

		output[nDst] = curVocabInd;
	}
}


/// Expose
extern "C"
void OHNN_CudaOneHotTemporalBowStack_updateOutput(
		THCState *state,
		// In
        THCudaTensor *input,
        double p,
        double padVocabInd,
        // Out
        THCudaTensor *output)
{
	DEBUG_PRINT(("in OHNN_CudaOneHotTemporalBowStack_updateOutput\n"));
	THAssert(THCudaTensor_checkGPU(state, 2, input, output));
	// TODO: arg check?

	// input: B, M (,V)
	// output: B, Mp, C
	int B = THCudaTensor_size(state, input, 0);
	int M = THCudaTensor_size(state, input, 1);
	int BM = B*M;
	int Mp = M*(int(p));
	DEBUG_PRINT(("B = %d, M = %d, p = %d, padVocabInd = %d\n", B, M, (int)p, (int)padVocabInd));

	// prepare data
	THCudaTensor_resize2d(state, output, B, Mp);

	// stack bow input
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 grid(DIV_CEIL(BM, CUDA_NUM_THREADS));
	dim3 block(CUDA_NUM_THREADS);
	temporal_bow_statck_kernelV1<<<grid, block, 0, stream>>>(
			THCudaTensor_data(state, input), B, M, (int)p, (int)padVocabInd,
			THCudaTensor_data(state, output)
	);

	// check error
	DEBUG_PRINT(("checking cuda error\n"));
	THCudaCheck(hipGetLastError());
	DEBUG_PRINT(("done, no cuda error\n"));

	DEBUG_PRINT(("leaving OHNN_CudaOneHotTemporalBowStack_updateOutput\n"));
}
