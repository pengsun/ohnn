#include "hip/hip_runtime.h"
#include "THC.h"
#include "common.h"
#include <cstdio>
#include <hipsparse.h>


/// Helper
// TODO: make it a template fun and move to a separate file
__global__ void temporal_bow_statck_kernelV2(
		float *input, int B, int M, int p, int padBegLen, int padEndLen, int padIndValue,
		float *output)
{
	// a Naive impl
	int Md = (M + padBegLen + padEndLen) - p + 1;

	int nWinDst = blockIdx.x * blockDim.x + threadIdx.x;
	if (nWinDst >= B*Md) return;

	int iBat = nWinDst / Md;
	int iWin = nWinDst % Md;
	for (int i = 0; i < p; ++i) { // scan each word in the window
		int iCurWord = iWin - padBegLen + i;
		int curVocabInd = padIndValue;
		int nDst = nWinDst*p + i;

		if (iCurWord >= 0 && iCurWord < M) { // inside the source iBat sequence
			curVocabInd = (int)input[iBat*M + iCurWord];

			if (curVocabInd != padIndValue) { // encounter a normal word
				// scan previous word in the same window (Warp divergence here?)
				for (int k = 1; k <= i; ++k) {
					if (curVocabInd == output[nDst-k]) { // encounter a duplicate
						curVocabInd = padIndValue;
						break;
					}
				}
			}
		}

		output[nDst] = curVocabInd;
	}
}

/// Expose
extern "C"
void OHNN_CudaOneHotTemporalBowStack_updateOutput(
		THCState *state,
		// In
        THCudaTensor *input,
        double p,
        double padBegLen,
        double padEndLen,
        double padIndValue,
        // Out
        THCudaTensor *output)
{
	DEBUG_PRINT(("in OHNN_CudaOneHotTemporalBowStack_updateOutput\n"));
	THAssert(THCudaTensor_checkGPU(state, 2, input, output));
	// TODO: arg check?

	// input: B, M (,V)
	// output: B, M'*p (,V)
	int B = THCudaTensor_size(state, input, 0);
	int M = THCudaTensor_size(state, input, 1);
	int Md = (M + (int)padBegLen + (int)padEndLen) - (int)p + 1; // output seq length
	int Mdp = Md*(int)p;
	int BMd = B*Md;
	DEBUG_PRINT(("B = %d, M = %d, Md = %d, p = %d\n", B, M, Md, (int)p));
	DEBUG_PRINT(("padBegLen = %d, padEndLen = %d, padVocabInd = %d\n", (int)padBegLen, (int)padEndLen, (int)padIndValue));
	DEBUG_PRINT(("Mdp = %d, BMd = %d\n", Mdp, BMd));

	// prepare data
	THCudaTensor_resize2d(state, output, B, Mdp);

	// stack bow input
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 grid(DIV_CEIL(BMd, CUDA_NUM_THREADS));
	dim3 block(CUDA_NUM_THREADS);
	temporal_bow_statck_kernelV2<<<grid, block, 0, stream>>>(
			THCudaTensor_data(state, input), B, M, (int)p,
			(int)padBegLen, (int)padEndLen, (int)padIndValue,
			THCudaTensor_data(state, output)
	);

	// check error
	DEBUG_PRINT(("checking cuda error\n"));
	THCudaCheck(hipGetLastError());
	DEBUG_PRINT(("done, no cuda error\n"));

	DEBUG_PRINT(("leaving OHNN_CudaOneHotTemporalBowStack_updateOutput\n"));
}
